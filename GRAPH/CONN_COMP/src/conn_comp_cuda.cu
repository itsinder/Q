#include "hip/hip_runtime.h"
extern "C" {
#include "q_incs.h"
#include "_mmap.h"
}


#define NODE_TYPE int32_t
#define MAXLINE 65535

__global__
static void
any_change(
    NODE_TYPE *lb,
    NODE_TYPE *ub,
    NODE_TYPE *to,
    NODE_TYPE *lbl,
    uint64_t n_nodes,
    bool *is_any_change
    )
{
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint64_t stride = blockDim.x * gridDim.x;
  for (uint64_t i = index; i < n_nodes; i += stride) {
    bool l_is_any_change = false;
    if ( ub[i] <= lb[i] ) { continue; }
    NODE_TYPE minval = lbl[i];
    for ( int64_t j = lb[i]; j < ub[i]; j++ ) {
      minval = mcr_min(minval, lbl[to[j]]);
    }
    if ( lbl[i] != minval ) {
      l_is_any_change = true;
      lbl[i] = minval;
    }

    if ( ( l_is_any_change ) && ( *is_any_change == false ) ) {
      *is_any_change = true;
    }

  }
}


int
main(
    int argc,
    char **argv
    )
{
  int status = 0;
  FILE *lbfp = NULL;
  FILE *ubfp = NULL;
  FILE *tofp = NULL;
  NODE_TYPE *lbl = NULL;
  NODE_TYPE *lb = NULL;
  NODE_TYPE *ub = NULL;
  NODE_TYPE *to = NULL;
  char *lb_X = NULL; size_t lb_nX = 0;
  char *ub_X = NULL; size_t ub_nX = 0;
  char *to_X = NULL; size_t to_nX = 0;

  if ( argc != 1 ) { status = -1; return status; }

  status = rs_mmap("lb.bin", &lb_X, &lb_nX, 0);
  lb = (NODE_TYPE *)lb_X;

  status = rs_mmap("ub.bin", &ub_X, &ub_nX, 0);
  ub = (NODE_TYPE *)ub_X;

  status = rs_mmap("to.bin", &to_X, &to_nX, 0);
  to = (NODE_TYPE *)to_X;

  uint64_t n_nodes = lb_nX / sizeof(NODE_TYPE);
  fprintf(stderr, "Working on  %ld nodes \n", n_nodes);

  // lbl = (NODE_TYPE *)malloc(n_nodes * sizeof(NODE_TYPE));
  hipMallocManaged(&lbl, n_nodes * sizeof(NODE_TYPE));
  // return_if_malloc_failed(lbl);
  if ( lbl == NULL ) { return -1; }
  for ( unsigned int i = 0; i < n_nodes; i++ ) {
    lbl[i] = i;
  }

  uint64_t blockSize = 256;
  uint64_t numBlocks = (n_nodes + 256 - 1) / blockSize;

  bool is_any_change = true; // just to get in the first tome
  for ( int iter = 0; is_any_change == true; iter++ ) {
    // any_change(lb, ub, to, lbl, n_nodes, &is_any_change);
    any_change<<<numBlocks, blockSize>>>(lb, ub, to, lbl, n_nodes, &is_any_change);
    hipDeviceSynchronize();

/*
#pragma omp parallel for schedule(static)
    for ( uint64_t i = 0; i < n_nodes; i++ ) {
      bool l_is_any_change = false;
      if ( ub[i] <= lb[i] ) { continue; }
      NODE_TYPE minval = lbl[i];
      for ( int64_t j = lb[i]; j < ub[i]; j++ ) {
        minval = mcr_min(minval, lbl[to[j]]);
      }
      if ( lbl[i] != minval ) {
        l_is_any_change = true;
        lbl[i] = minval;
      }
      if ( ( l_is_any_change ) && ( is_any_change == false ) ) {
        is_any_change = true;
      }
    }
*/
    fprintf(stderr, "Pass %d \n", iter);
  }
  hipFree(lbl);
  return status;

//BYE:
  /*
  if ( lb_X != NULL ) { munmap(lb_X, lb_nX); }
  if ( ub_X != NULL ) { munmap(ub_X, lb_nX); }
  if ( to_X != NULL ) { munmap(to_X, lb_nX); }
  fclose_if_non_null(lbfp);
  fclose_if_non_null(ubfp);
  fclose_if_non_null(tofp);
  */
  //hipFree(lbl);
  //return status;
}

