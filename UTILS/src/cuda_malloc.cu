extern "C" {
//START_INCLUDES
#include "q_incs.h"
//STOP_INCLUDES
#include "_cuda_malloc.h"
}

//START_FUNC_DECL
void *
cuda_malloc(
    int64_t N
    )
//STOP_FUNC_DECL
{
  // CUDA: malloc using hipMallocManaged
  static void *ptr;
  hipMallocManaged(&ptr, N);
  return ptr;
}
