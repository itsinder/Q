extern "C" {
//START_INCLUDES
#include "q_incs.h"
//STOP_INCLUDES
#include "_cuda_malloc.h"
}

//START_FUNC_DECL
int 
cuda_malloc(
    int64_t N,
    void *ptr
    )
//STOP_FUNC_DECL 
{
  int status = 0;
  hipMallocManaged(&ptr, N);
  return status;
}
