extern "C" {
//START_INCLUDES
#include "q_incs.h"
//STOP_INCLUDES
}
//START_FUNC_DECL
int 
cuda_malloc(
    int64_t N,
    void *ptr
    )
//STOP_FUNC_DECL 
{
  int status = 0;
  hipMallocManaged(&ptr, N);
  return status;
}
